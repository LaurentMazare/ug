
#include <hip/hip_runtime.h>
using uint32_t = unsigned int;

template <typename T>
__device__ void rope(const T * src, const T * cos, const T * sin, T * dst, const uint32_t bh, const uint32_t td, const uint32_t d) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (2 * idx >= bh * td) return;

    uint32_t i_bh = idx / (td / 2);
    uint32_t i_td = idx - (td / 2) * i_bh;
    uint32_t i_t = i_td / (d / 2);
    uint32_t i_d = i_td - (d / 2) * i_t;
    uint32_t i1 = i_bh * td + i_t * d + i_d;
    uint32_t i2 = i1 + d / 2;
    uint32_t i_cs = i_t * (d / 2) + i_d;
    T c = cos[i_cs];
    T s = sin[i_cs];

    dst[i1] = src[i1] * c - src[i2] * s;
    dst[i2] = src[i1] * s + src[i2] * c;
}

extern "C" __global__ void rope_f32(
    const float * src,
    const float * cos,
    const float * sin,
    float * dst,
    const uint32_t bh,
    const uint32_t td,
    const uint32_t d
) {
  rope<float>(src, cos, sin, dst, bh, td, d);
}
